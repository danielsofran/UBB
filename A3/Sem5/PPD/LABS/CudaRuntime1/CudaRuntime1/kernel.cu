#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <chrono>

#include "utils.h"

using namespace std;

// 1 row of 10 blocks each having 10 threads each computing 10 lines
#define LINES_PER_THREAD 10
#define BLOCK_COUNT 10
#define BLOCK_DIM 10
#define N 1000
#define EXTENDED_N_SIZE 

#define INDEX_X 0
#define INDEX_Y 0

__device__ inline int* getRow(int* dev, size_t pitch, int i) {
    return (int*)((char*)dev + i * pitch);
}

__device__ int *getElem(int *dev, size_t pitch, int i, int j) {
    int* row = getRow(dev, pitch, i);
    return &row[j];
}

__device__ void printMat(int *dev, size_t pitch) {
    int n = 3;
    for (int i = 0; i < n; ++i, printf("\n"))
        for (int j = 0; j < n; ++j)
            printf("%d ", *getElem(dev, pitch, i + INDEX_X, j + INDEX_Y));
}

__host__ void printMatHost(int **host) {
    int n = 3;
    for (int i = 0; i < n; ++i, printf("\n"))
        for (int j = 0; j < n; ++j)
            printf("%d ", host[i + INDEX_X][j + INDEX_Y]);
}

__global__ void convolution(int *devA, size_t pitchA, int *devC, size_t pitchC) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int lineStart = id * 10 + 1, lineEnd = lineStart + 10;
    //printMat(devA, pitchA);
    //printf("%d", *getElem(devA, pitchA, 1000, 0));
    //int* elem = getElem(devA, pitchA, 0, 0);
    //*elem = 0;

    //copy frontiers

    int current[N + 2],
        above[N + 2],
        frontierAbove[N + 2],
        frontierBelow[N + 2];

    //copy(getElem(devA, pitchA, lineStart - 1, 0), getElem(devA, pitchA, lineStart - 1, N + 2), frontierAbove);
    for (int i = 0; i < N + 2; ++i) {
        frontierAbove[i] = *getElem(devA, pitchA, lineStart - 1, i);
        frontierBelow[i] = *getElem(devA, pitchA, lineEnd, i);
    }
    __syncthreads(); // Wait for all threads to get their frontiers

    //hipMemcpyAsync(&above, getRow(devA, pitchA, lineStart), EXTENDED_N_SIZE, hipMemcpyDeviceToDevice);
    for (int i = 0; i < N + 2; ++i)
        above[i] = *getElem(devA, pitchA, lineStart, i);
    int left = (*getElem(devA, pitchA, lineStart, 0));

    // first line
    for (int j = 1; j <= N; j++) {
        int value = 0;
        for (int ii = 0; ii <= 1; ii++) {
            for (int jj = -1; jj <= 1; jj++) {
                if (ii == 0 && jj == -1) {
                    value += left * (*getElem(devC, pitchC, ii + 1, jj + 1));
                } else {
                    value += (*getElem(devA, pitchA, lineStart + ii, j + jj)) * (*getElem(devC, pitchC, ii + 1, jj + 1));
                }
            }
        }
        for (int jj = -1; jj <= 1; jj++) {
            value += frontierAbove[j + jj] * (*getElem(devC, pitchC, 0, jj + 1));
        }
        left = (*getElem(devA, pitchA, lineStart, j));
        int *elemToSet = getElem(devA, pitchA, lineStart, j);
        *elemToSet = value;
    }
    left = (*getElem(devA, pitchA, lineStart + 1, 0));

    //inner lines
    for (int i = lineStart + 1; i < lineEnd - 1; i++) {
        //hipMemcpy(&current, getRow(devA, pitchA, lineStart), EXTENDED_N_SIZE, hipMemcpyDeviceToDevice);
        for (int k = 0; k < N + 2; ++k)
            current[k] = *getElem(devA, pitchA, i, k);

        for (int j = 1; j <= N; j++) {
            int value = 0;
            for (int ii = 0; ii <= 1; ii++) {
                for (int jj = -1; jj <= 1; jj++) {
                    if (ii == 0 && jj == -1) {
                        value += left * (*getElem(devC, pitchC, ii + 1, jj + 1));
                    } else {
                        value += (*getElem(devA, pitchA, i + ii, j + jj)) * (*getElem(devC, pitchC, ii + 1, jj + 1));
                    }
                }
            }
            for (int jj = -1; jj <= 1; jj++) {
                value += above[j + jj] * (*getElem(devC, pitchC, 0, jj + 1));
            }
            if (j == N) {
                left = (*getElem(devA, pitchA, i + 1, 0));
            } else {
                left = (*getElem(devA, pitchA, i, j));
            }
            *getElem(devA, pitchA, i, j) = value;
        }
        //hipMemcpy(&above, &current, EXTENDED_N_SIZE, hipMemcpyDeviceToDevice);
        for (int k = 0; k < N + 2; ++k)
            above[k] = current[k];
    }

    //last line
    for (int j = 1; j <= N; j++) {
        int value = 0;
        for (int jj = -1; jj <= 1; jj++) {
            value += above[j + jj] * (*getElem(devC, pitchC, 0, jj + 1));
        }
        for (int jj = -1; jj <= 1; jj++) {
            if (jj == -1) {
                value += left * (*getElem(devC, pitchC, 1, jj + 1));
            } else {
                value += (*getElem(devA, pitchA, lineEnd - 1, j + jj)) * (*getElem(devC, pitchC, 1, jj + 1));
            }
        }
        for (int jj = -1; jj <= 1; jj++) {
            value += frontierBelow[j + jj] * (*getElem(devC, pitchC, 2, jj + 1));
        }
        left = (*getElem(devA, pitchA, lineEnd - 1, j));
        *getElem(devA, pitchA, lineEnd - 1, j) = value;
    }
}

int main() {
    std::chrono::steady_clock::time_point t_start, t_end;
    hipSetDevice(0);
    const string filename = "..\\x64\\Debug\\data.txt";
    int **hostA, **hostC;
    int *deviceA, *deviceC;
    size_t pitchA, pitchC;

    // initialize host memory
    hostA = (int **) malloc(sizeof(int *) * (N + 2));
    for (int i = 0; i < N + 2; ++i) hostA[i] = (int *) malloc(sizeof(int) * (N + 2));
    hostC = (int **) malloc(sizeof(int *) * 3);
    for (int i = 0; i < 3; ++i) hostC[i] = (int *) malloc(sizeof(int) * 3);

    // initialize device (GPU) memory
    hipMallocPitch(&deviceA, &pitchA, (N + 2) * sizeof(int), N + 2);
    hipMallocPitch(&deviceC, &pitchC, 3 * sizeof(int), 3);

    readMatrixFromFile(filename, N, N, hostA, hostC);
    borderMatrix(hostA, N, N);

    // Copy data from host to device for matrix A
    for (int i = 0; i < N + 2; ++i) {
        hipMemcpy(reinterpret_cast<char *>(deviceA) + i * pitchA, hostA[i], (N + 2) * sizeof(int),
                   hipMemcpyHostToDevice);
    }

    // Copy data from host to device for matrix C
    for (int i = 0; i < 3; ++i) {
        hipMemcpy(reinterpret_cast<char *>(deviceC) + i * pitchC, hostC[i], 3 * sizeof(int), hipMemcpyHostToDevice);
    }

    t_start = std::chrono::high_resolution_clock::now();

    convolution<<<BLOCK_COUNT, BLOCK_DIM>>>(deviceA, pitchA, deviceC, pitchC);
    hipDeviceSynchronize();

    t_end = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < N + 2; ++i) {
        hipMemcpy(reinterpret_cast<char *>(hostA[i]), reinterpret_cast<char *>(deviceA) + i * pitchA,
            (N + 2) * sizeof(int), hipMemcpyDeviceToHost);
    }

    writeMatrixToFile("..\\x64\\Debug\\outThreads.txt", hostA, N + 2, N + 2);

    hipFree(deviceA);
    hipFree(deviceC);
    for (int i = 0; i < N + 2; ++i)
        free(hostA[i]);
    free(hostA);
    for (int i = 0; i < 3; ++i)
        free(hostC[i]);
    free(hostC);

    hipError_t cudaStatus = hipSuccess;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    double elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();

    cout << elapsed_time_ms;

    return 0;
}

